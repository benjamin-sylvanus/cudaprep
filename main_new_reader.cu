#include "hip/hip_runtime.h"
#include "./src/newsimreader.h"
#include "./src/simreader.h"
#include "./src/simulation.h"
#include "./src/controller.h"
#include "./src/viewer.h"
#include "./src/funcs.h"
#include "./src/overloads.h"
#include "vector"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <cstring>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <chrono>
#include <thread>
#include <hipblas.h>

#define HIP_PI_F 3.141592654f
#define PI 3.14159265358979323846
#define LDPI 3.141592653589793238462643383279502884L
#define timepoints 1000
#define Nc 2
#define Nbvec 3
#define SOD (sizeof(double))
#define SOI (sizeof(int))
#define SOF (sizeof(float))
#define SOD3 (sizeof(double3))
#define SOD4 (sizeof(double4))
#define SOI3 (sizeof(int3))
#define SOI4 (sizeof(int4))
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

using std::cin;
using std::cout;
using std::endl;
using std::chrono::duration;
using std::chrono::duration_cast;
using std::chrono::high_resolution_clock;
using std::chrono::milliseconds;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

/**
 * @brief Initializes the random number generator
 * @param state pointer to the random number generator
 * @param seed seed for the random number generator
 */
__global__ void setup_kernel(hiprandStatePhilox4_32_10_t *state, unsigned long seed)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

/**
 * @brief determine the volume fraction via random sampling over n particles
 * @param state
 * @param Bounds
 * @param d4swc
 * @param nlut
 * @param NewIndex
 * @param IndexSize
 * @param n
 * @param label
 * @param vf
 */
__global__ void volfrac(hiprandStatePhilox4_32_10_t *state, int3 Bounds, double4 *d4swc, int *nlut, int *NewIndex, int3 IndexSize, int n, int *label, double *vf)
{
    int gid = threadIdx.x + blockDim.x * blockIdx.x;
    int N = 10000;
    if (gid == 0)
    {
        printf("ok %d\n", n);
    }
    if (gid < N)
    {
        label[gid] = 0;
        int R = 0;
        double3 nextpos;
        int3 upper;
        int3 lower;
        int3 floorpos;
        int3 b_int3 = make_int3(Bounds.x, Bounds.y, Bounds.z);
        int3 i_int3 = make_int3(IndexSize.x, IndexSize.y, IndexSize.z);
        double3 b_d3 = make_double3((double)b_int3.x, (double)b_int3.y, (double)b_int3.z);
        hiprandStatePhilox4_32_10_t localstate = state[gid];
        double4 xr;
        double3 A;
        bool cont = true;

        // init local state var
        xr = hiprand_uniform4_double(&localstate);

        // set particle initial position
        A = make_double3(xr.x * b_d3.x, xr.y * b_d3.y, xr.z * b_d3.z);

        nextpos = A;

        // floor of position -> check voxels
        floorpos.x = (int)A.x;
        floorpos.y = (int)A.y;
        floorpos.z = (int)A.z;

        // upper bounds of lookup table
        upper.x = floorpos.x < b_int3.x;
        upper.y = floorpos.y < b_int3.y;
        upper.z = floorpos.z < b_int3.z;

        // lower bounds of lookup table
        lower.x = floorpos.x > 0;
        lower.y = floorpos.y > 0;
        lower.z = floorpos.z > 0;

        double4 parent;
        double4 child;
        int2 vindex;
        int id_test = s2i(floorpos, Bounds);
        int test_lutvalue = nlut[id_test];
        double dist2;

        // loop over connections
        for (int page = 0; page < i_int3.z; page++)
        {
            int3 c_new = make_int3(test_lutvalue, 0, page);
            int3 p_new = make_int3(test_lutvalue, 1, page);
            vindex.x = NewIndex[s2i(c_new, i_int3)] - 1;
            vindex.y = NewIndex[s2i(p_new, i_int3)] - 1;
            // if theres a index
            if ((vindex.x) != -1)
            {
                child = make_double4(d4swc[vindex.x].x, d4swc[vindex.x].y, d4swc[vindex.x].z, d4swc[vindex.x].w);
                parent = make_double4(d4swc[vindex.y].x, d4swc[vindex.y].y, d4swc[vindex.y].z, d4swc[vindex.y].w);

                // distance squared between child parent
                dist2 = distance2(parent, child);

                // determine whether particle is inside this connection
                bool inside = swc2v(nextpos, child, parent, dist2);

                // if it is inside the connection we don't need to check the remaining.
                if (inside)
                {
                    // update the particles state
                    label[gid] = 1;
                    // printf("It was inside\n");
                    // end for p loop
                    page = i_int3.z;
                }
            }
        }
    }

    __syncthreads(); // Correct synchronization function

    // sum the particles inside and calculate volume fraction
    if (gid == 0)
    {
        printf("gid %d\n", gid);

        double R = 0;
        for (int i = 0; i < N; i++)
        {
            R += (double)label[i];
        }

        printf("R: %.0f\n", R);
        printf("R/N: %.0f/%d = %.4f\n", R, N, R / (double)N);

        vf[0] = (double)R / (double)N; // Update the volume fraction using pointers
    }
}

/**
 * @brief Simulation Kernel for the GPU
 * @param savedata - the data to be saved
 * @param dx2 - the second moment of the diffusion tensor
 * @param dx4 - the fourth moment of the diffusion tensor
 * @param Bounds - the bounds of the simulation
 * @param state - the random number generator state
 * @param SimulationParams - the simulation parameters
 * @param d4swc - the swc data
 * @param nlut - the neighbor lookup table
 * @param NewIndex - the new index
 * @param IndexSize - the index size
 * @param size - the number of particles
 * @param iter - the number of iterations
 * @param debug - whether or not to print debug statements
 * @param point - the point to simulate
 * @param SaveAll - whether or not to save all data
 * @param Reflections - the reflections
 * @param Uref - the unreflected data
 * @param flip - the flip data
 * @param T2 - the T2 data
 * @param T - the T data
 * @param Sig0 - the Sig0 data
 * @param SigRe - the SigRe data
 * @param BVec - the BVec data
 * @param BVal - the BVal data
 * @param TD - the TD data
 */
__global__ void simulate(double *savedata, double *dx2, double *dx4, int3 Bounds, hiprandStatePhilox4_32_10_t *state,
                         double *SimulationParams,
                         double4 *d4swc, int *nlut, int *NewIndex, int3 IndexSize, int size, int iter, bool debug,
                         double3 point, int SaveAll, double *Reflections, double *Uref, int *flip,
                         double *T2, double *T, double *Sig0, double *SigRe, double *BVec, double *BVal, double *TD)
{

    int gid = threadIdx.x + blockDim.x * blockIdx.x;
    if (gid < size)
    {
        /** TODO: Verify this order is still correct
            @index particle_num = SimulationParams[0]
            @index step_num = SimulationParams[1]
            @index step_size = SimulationParams[2]
            @index perm_prob = SimulationParams[3]
            @index init_in = SimulationParams[4]
            @index D0 = SimulationParams[5]
            @index d = SimulationParams[6]
            @index scale = SimulationParams[7]
            @index tstep = SimulationParams[8]
        */
        double step_size = SimulationParams[2];
        double perm_prob = SimulationParams[3];
        int init_in = (int)SimulationParams[4];
        double tstep = SimulationParams[8];
        double vsize = SimulationParams[9];
        double3 A;
        int2 parstate;
        double4 xi;
        double3 nextpos;
        double3 xnot;
        int3 upper;
        int3 lower;
        int3 floorpos;
        int Tstep = iter / timepoints;
        double fstep = 1;
        double pi = PI;

        int3 b_int3 = make_int3(Bounds.x, Bounds.y, Bounds.z);
        int3 i_int3 = make_int3(IndexSize.x, IndexSize.y, IndexSize.z);

        double _T2[Nc];
        double t2[3] = {80, 40, 60};
        for (int j = 0; j < Nc; j++)
        {
            _T2[j] = t2[j];
        }

        double3 d2 = make_double3(0.0, 0.0, 0.0);
        bool completes;
        bool flag;
        double step = step_size;

        // init local state var
        hiprandStatePhilox4_32_10_t localstate = state[gid];
        xi = hiprand_uniform4_double(&localstate);

        // prob to accept the sampling: init in: 4. per compartment. both <= to 1.  one should be 1
        A = initPosition(gid, dx2, Bounds, state, SimulationParams, d4swc, nlut, NewIndex, IndexSize,
                         size, iter, init_in, debug, point); // initialize position inside cell

        xnot = make_double3(A.x, A.y, A.z); // record initial position
        flag = false;                       // flag is initially false

        parstate = make_int2(1, 1); // particle state [previous step, current step]
        int parlut = 1;             // particle within bounds of LUT?
        double t[Nc] = {0};         // add tstep for step in compartment

        for (int i = 0; i < iter; i++)
        {
            xi = hiprand_uniform4_double(&localstate); // generate uniform randoms for step
            completes = xi.w < perm_prob;             // determine if step executes
            computeNext(A, step, xi, nextpos, pi);    // compute the next position

            // check coordinate validity
            validCoord(nextpos, A, b_int3, upper, lower, floorpos, Reflections, Uref, gid, i, size, iter, flip);
            floorpos = make_int3((int)nextpos.x, (int)nextpos.y, (int)nextpos.z);
            int test_lutvalue = nlut[s2i(floorpos, b_int3)];
            bool inside = checkConnections(i_int3, test_lutvalue, nextpos, NewIndex, d4swc, fstep); // check if particle inside
            parstate.y = (inside) ? 1 : 0;

            /**
            * @cases particle inside? 0 0 - update
            * @cases particle inside? 0 1 - check if updates
            * @cases particle inside? 1 0 - check if updates
            * @cases particle inside? 1 1 - update

            * @vars:
            * t[0]: inside
            * t[1]: outside
            */

            // particle inside: [0 0] || [1 1]
            if (parstate.x == parstate.y)
            {
                A = nextpos;
                if (parstate.x)
                {
                    t[0] = t[0] + tstep;
                }
                else
                {
                    t[1] = t[1] + tstep;
                }
            }

            // particle inside: [1 0]
            if (parstate.x && !parstate.y)
            {
                if (!completes)
                {
                    t[0] = t[0] + tstep;
                }
                else
                {
                    A = nextpos;
                    parstate.x = parstate.y;
                    t[0] = t[0] + tstep * fstep;
                    t[1] = t[1] + tstep * (1 - fstep);
                }
            }

            // particle inside [0 1]
            if (!parstate.x && parstate.y)
            {
                if (!completes)
                {
                    t[1] = t[1] + tstep;
                }
                else
                {
                    A = nextpos;
                    parstate.x = parstate.y;
                    t[0] = t[0] + tstep * (1 - fstep);
                    t[1] = t[1] + tstep * fstep;
                }
            }

            // Store Position Data
            if (SaveAll)
            {
                int3 dix = make_int3(size, iter, 3);
                int3 did[4];
                did[0] = make_int3(gid, i, 0);
                did[1] = make_int3(gid, i, 1);
                did[2] = make_int3(gid, i, 2);
                did[3] = make_int3(s2i(did[0], dix), s2i(did[1], dix), s2i(did[2], dix));
                set(savedata, did[3], A);
            }
            // Store Tensor Data
            {
                diffusionTensor(&A, &xnot, vsize, dx2, dx4, &d2, i, gid, iter, size);
                // https://github.com/NYU-DiffusionMRI/monte-carlo-simulation-3D-RMS/blob/master/part1_demo3_simulation.m
            }

            // Store Signal Data
            {
                if (i % Tstep == 0)
                {
                    int tidx = i / Tstep;
                    // loop over compartments
                    double s0 = 0.0;
                    for (int j = 0; j < 2; j++)
                    {
                        /**
                         * @var s0 is our summation variable
                         * @var t[j] is the time in compartment j
                         * @var T2 is the T2 Relaxation in Compartment j
                         */
                        s0 = s0 + (double)(t[j] / _T2[j]); // TODO implement "t" as time in each compartment
                    }

                    s0 = exp(-1.0 * s0);
                    atomicAdd(&Sig0[tidx], s0);

                    // // Signal
                    // for(int j = 0; j < Nbvec; j++)
                    // {
                    //     // access b value and b vector
                    //     double bval = bvalues[j];
                    //     double3 bvec =  bvectors[j];
                    //     td = TD[tidx];
                    //     qx = sqrt(b.w/td) * dot(d2,bvec)
                    //     atomicAdd(&sigRe[Nbvec * tidx + j], s0 * cos(qx));
                    // }

                    for (int j = 0; j < Nc; j++)
                    {
                        t[j] = 0;
                    }
                }
            }
        }
    }
}

int main(int argc, char *argv[])
{
    hipEvent_t start_c, stop_c;
    hipEventCreate(&start_c);
    hipEventCreate(&stop_c);
    float milliseconds = 0;
    int NC = 2;
    system("clear");
    int SaveAll = 1;
    std::string root;
    // Parse config path from arguments
    if (argc < 2) {
        root = "data/simulation_configs/test";
    } else {
        root = argv[1];
    }

    const std::string binaryFile = root + "/simulation_config.bin";
    const std::string jsonFile = root + "/simulation_config.json";

    std::vector<Variable> variables = NewSimReader::parseJson(jsonFile);
    NewSimReader::readBinaryFile(binaryFile, variables);

    // Initialize variables to hold extracted data
    uint64_t particleNum = 0, stepNum = 0;
    double stepSize = 0, permProb = 0;
    double initIn = 0, D0 = 0, d = 0, scale = 0, tstep = 0, vsize = 0;
    double *swcmat = nullptr;
    uint64_t *LUT = nullptr;
    uint64_t *C = nullptr;
    uint64_t *pairs = nullptr;
    uint64_t *boundSize = nullptr;

    // Extract data from variables
    NewSimReader::extractData(variables, particleNum, stepNum, stepSize, permProb,
                            initIn, D0, d, scale, tstep, vsize,
                            swcmat, LUT, C, pairs, boundSize, true);
    NewSimReader::previewConfig(variables, particleNum, stepNum, stepSize, permProb, initIn, D0, d, scale, tstep, vsize,
                            swcmat, LUT, C, pairs, boundSize, true);


    // Get array dimensions from variables
    std::vector<std::vector<int>> arrdims;
    std::vector<int> swc_dims, lut_dims, index_dims, pairs_dims, bounds_dims;

    for (const auto& var : variables) {
        if (var.name == "swcmat") {
            swc_dims = var.size;           
        }
        else if (var.name == "LUT") lut_dims = var.size;
        else if (var.name == "C") index_dims = var.size;
        else if (var.name == "pairs") pairs_dims = var.size;
        else if (var.name == "bounds") bounds_dims = var.size;
    }

    arrdims = {swc_dims, lut_dims, index_dims, pairs_dims, bounds_dims};    // Calculate new index size
    int newindexsize = index_dims[0] * index_dims[1] * index_dims[2];
    int LUTSIZE = lut_dims[0] * lut_dims[1] * lut_dims[2];
    // Convert to simulation parameters
    int size = static_cast<int>(particleNum);
    int iter = static_cast<int>(stepNum);
    double simparam[10] = {
        static_cast<double>(size),
        static_cast<double>(iter),
        stepSize, permProb, initIn,
        D0, d, scale, tstep, vsize
    };

    // Get dimensions from boundSize
    int3 bounds = make_int3(boundSize[0], boundSize[1], boundSize[2]);
    int prod = bounds.x * bounds.y * bounds.z;

    // Get nrow from swcmat dimensions
    size_t nrow = swc_dims[0];  // Now we can use swc_dims directly

    // Convert swcmat to r_swc format
    std::vector<double> r_swc(nrow * 6);
    for (int i = 0; i < nrow; i++) {
        for (int j = 0; j < 6; j++) {
            r_swc[i + nrow * j] = swcmat[i * 6 + j];
        }
    }
    // Create double4 array for swc_trim
    double4 swc_trim[nrow];
    double w_swc[nrow * 4];
    for (int i = 0; i < nrow; i++) {
        swc_trim[i].x = swcmat[i + nrow * 1];
        swc_trim[i].y = swcmat[i + nrow * 2];
        swc_trim[i].z = swcmat[i + nrow * 3];
        swc_trim[i].w = swcmat[i + nrow * 4];
    }
    for (int i = 0; i < nrow; i++) {
        w_swc[4 * i + 0] = swc_trim[i].x;
        w_swc[4 * i + 1] = swc_trim[i].y;
        w_swc[4 * i + 2] = swc_trim[i].z;
        w_swc[4 * i + 3] = swc_trim[i].w;
    }
    int block_size = 256;
    dim3 block(block_size);
    dim3 grid((size / block.x) + 1);
    size_t sa_size = (SaveAll) ? 
    static_cast<size_t>(particleNum) * static_cast<size_t>(stepNum) : 1;
    std::vector<uint64_t> lut(LUT, LUT + LUTSIZE);
    std::vector<uint64_t> indexarr(C,C + newindexsize);

    /// Unified Memory
    // Declare Unified Memory Pointers
    double4 *u_D4Swc;
    double *u_dx2, *u_dx4, *u_SimP, *u_T2, *u_T, *u_SigRe, *u_Sig0, *u_bvec,
        *u_bval, *u_TD, *mdx2, *mdx4, *u_AllData, *u_Reflections, *u_uref;

    int *u_NewLut, *u_NewIndex, *u_Flip;


    // new
    int *u_label;
    double *u_vf;
    int n = 10000000;
    hipMallocManaged(&u_D4Swc, nrow * SOD4);
    hipMallocManaged(&u_dx2, 6 * iter * SOD);
    hipMallocManaged(&u_dx4, 15 * iter * SOD);
    hipMallocManaged(&u_SimP, 10 * SOD);
    hipMallocManaged(&mdx2, 6 * iter * SOD);
    hipMallocManaged(&mdx4, 15 * iter * SOD);
    hipMallocManaged(&u_AllData, 3 * sa_size * SOD);
    hipMallocManaged(&u_Reflections, 3 * iter * size * SOD);
    hipMallocManaged(&u_uref, 3 * iter * size * SOD);
    hipMallocManaged(&u_T2, Nc * SOD);
    hipMallocManaged(&u_T, Nc * SOD);
    hipMallocManaged(&u_SigRe, Nbvec * timepoints * SOD);
    hipMallocManaged(&u_Sig0, timepoints * SOD);
    hipMallocManaged(&u_bvec, Nbvec * 3 * SOD);
    hipMallocManaged(&u_bval, Nbvec * SOD);
    hipMallocManaged(&u_TD, Nbvec * SOD);
    hipMallocManaged(&u_NewLut, prod * SOI);
    hipMallocManaged(&u_NewIndex, newindexsize * SOI);
    hipMallocManaged(&u_Flip, 3 * size * SOI);

    hipMallocManaged(&u_label, n * SOI);
    hipMallocManaged(&u_vf, SOD);
    //
    printf("Allocated Host Data\n");

    // Call Function to Set the Values for Host
    setup_data(u_dx2, u_dx4, u_SimP, u_D4Swc, u_NewLut, u_NewIndex, u_Flip, simparam, swc_trim, mdx2, mdx4,
               u_AllData, u_Reflections, u_uref, u_T2, u_T, u_SigRe, u_Sig0, u_bvec, u_bval, u_TD,
               lut, indexarr, bounds, size, iter, nrow, prod, newindexsize, sa_size, Nbvec, timepoints, NC, n, u_vf, u_label);
    // option for printing in kernel
    bool debug = false;
    double3 point = make_double3(u_D4Swc[0].x, u_D4Swc[0].y, u_D4Swc[0].z);
    // Create Random State Pointer Pointers
    hiprandStatePhilox4_32_10_t *deviceState;

    hipEventRecord(start_c);
    // Allocate Memory on Device
    gpuErrchk(hipMalloc((hiprandStatePhilox4_32_10_t **)&deviceState, size * sizeof(hiprandStatePhilox4_32_10_t)));

    // Set Values for Device
    setup_kernel<<<grid, block>>>(deviceState, 1); // initialize the random states



    int3 u_Bounds = make_int3(bounds.x, bounds.y, bounds.z);
    int3 u_IndexSize = make_int3(index_dims[0], index_dims[1], index_dims[2]);

    // Prefetch data asynchronously
    hipMemPrefetchAsync(&u_D4Swc, nrow * SOD4, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_dx2, 6 * iter * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_dx4, 15 * iter * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_SimP, 10 * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&mdx2, 6 * iter * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&mdx4, 15 * iter * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_AllData, 3 * sa_size * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_Reflections, 3 * iter * size * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_uref, 3 * iter * size * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_T2, Nc * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_T, Nc * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_SigRe, Nbvec * timepoints * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_Sig0, timepoints * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_bvec, Nbvec * 3 * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_bval, Nbvec * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_TD, Nbvec * SOD, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_NewLut, prod * SOI, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_NewIndex, newindexsize * SOI, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_Flip, 3 * size * SOI, hipCpuDeviceId);

    // new
    hipMemPrefetchAsync(&u_label, n * SOI, hipCpuDeviceId);
    hipMemPrefetchAsync(&u_vf, SOD, hipCpuDeviceId);

    // determine the volume fraction
    volfrac<<<grid, block>>>(deviceState, u_Bounds, u_D4Swc, u_NewLut, u_NewIndex, u_IndexSize, n, u_label, u_vf);
    hipEventRecord(stop_c);
    hipDeviceSynchronize();
    hipEventSynchronize(stop_c);
    hipEventElapsedTime(&milliseconds, start_c, stop_c);
    printf("Volume Fraction Kernel took %f seconds\n", milliseconds / 1e3);
    printf("Volume Fraction: %f\n", u_vf[0]);
    printf("Freeing Device Data\n");
    // Free Device Memory
    gpuErrchk(hipFree(u_label));
    gpuErrchk(hipFree(u_vf));

    /**
     * Call Kernel
     */
    {
        printf("Simulating...\n");
        simulate<<<grid, block>>>(u_AllData, u_dx2, u_dx4, u_Bounds, deviceState, u_SimP,
                                  u_D4Swc, u_NewLut, u_NewIndex, u_IndexSize, size,
                                  iter, debug, point, SaveAll, u_Reflections, u_uref, u_Flip,
                                  u_T2, u_T, u_Sig0, u_SigRe, u_bvec, u_bval, u_TD);
        hipEventRecord(stop_c);
    }

    // Wait for results
    hipDeviceSynchronize();

    /**
     * Copy Results From Device to Host
     */
    printf("Copying back to Host\n");
    hipEventSynchronize(stop_c);
    hipEventElapsedTime(&milliseconds, start_c, stop_c);
    clock_t end = clock();
    printf("Kernel took %f seconds\n", milliseconds / 1e3);
    printf("Writing results: ");

    auto t1 = high_resolution_clock::now();
    // // Write Results
    // {
    //     std::string outpath = root + "/results";
    //     writeResults(w_swc, u_SimP, u_dx2, mdx2, u_dx4, mdx4, u_T, u_Reflections, u_uref, u_Sig0, u_SigRe, u_AllData, iter, size, nrow, timepoints, Nbvec, sa_size, SaveAll, outpath, binaryFile, jsonFile);
    // }
    // Write New Results
    {   
        std::string outpath = root + "/results_new";
        writeResults_new(w_swc, u_SimP, u_dx2, mdx2, u_dx4, mdx4, u_T, u_Reflections, u_uref, u_Sig0, u_SigRe, u_AllData, iter, size, nrow, timepoints, Nbvec, sa_size, SaveAll, outpath, binaryFile, jsonFile);
    }

    auto t2 = high_resolution_clock::now();
    duration<double, std::milli> ms_double = t2 - t1;
    printf("%f seconds\n", ms_double.count() / 1e3);

    // Free Memory
    {
        printf("Freeing Memory: ");
        gpuErrchk(hipFree(u_dx2));
        gpuErrchk(hipFree(u_dx4));
        gpuErrchk(hipFree(u_SimP));
        gpuErrchk(hipFree(u_D4Swc));
        gpuErrchk(hipFree(u_NewLut));
        gpuErrchk(hipFree(u_NewIndex));
        gpuErrchk(hipFree(u_AllData));

        // Reflection Variables
        gpuErrchk(hipFree(u_Reflections));
        gpuErrchk(hipFree(u_uref));
        gpuErrchk(hipFree(u_Flip));

        // Signal Variables
        gpuErrchk(hipFree(u_T2));
        gpuErrchk(hipFree(u_T));
        gpuErrchk(hipFree(u_SigRe));
        gpuErrchk(hipFree(u_Sig0));
        gpuErrchk(hipFree(u_bvec));
        gpuErrchk(hipFree(u_bval));
        gpuErrchk(hipFree(u_TD));
        gpuErrchk(hipFree(deviceState));
    }

    printf("Done!\n");
    return 0;
}
