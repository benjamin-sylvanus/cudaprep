#include "hip/hip_runtime.h"
#include "./src/simreader.h"
#include "./src/simulation.h"
#include "./src/controller.h"
#include "./src/viewer.h"
#include "./src/funcs.h"
#include "vector"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <cstring>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <chrono>
#include <thread>
#include <hipblas.h>

#define HIP_PI_F 3.141592654f
#define PI 3.14159265358979323846
#define LDPI 3.141592653589793238462643383279502884L
using std::cout;
using std::cin;
using std::endl;
using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;

__global__ void setup_kernel(hiprandStatePhilox4_32_10_t *state, unsigned long seed) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}


__device__ bool checkConnections(int3 i_int3, int test_lutvalue, double3 nextpos, int* NewIndex, double4* d4swc) {
    int3 vindex;
    double4 child, parent;
    double dist2;

    // for each connection check if particle inside
    for (int page = 0; page < i_int3.z; page++) {

        // create a subscript indices
        int3 c_new = make_int3(test_lutvalue, 0, page);
        int3 p_new = make_int3(test_lutvalue, 1, page);

        // convert subscripted index to linear index and get value from Index Array
        vindex.x = NewIndex[s2i(c_new, i_int3)] - 1;
        vindex.y = NewIndex[s2i(p_new, i_int3)] - 1;

        if ((vindex.x) != -1) {
            //extract child parent values from swc
            child = d4swc[vindex.x];
            parent = d4swc[vindex.y];

            // calculate euclidean distance
            dist2 = pow(parent.x - child.x, 2) + pow(parent.y - child.y, 2) +
                    pow(parent.z - child.z, 2);

            // determine whether particle is inside this connection
            bool inside = swc2v(nextpos, child, parent, dist2);

            // if it is inside the connection we don't need to check the remaining.
            if (inside) {
                // end for p loop
                return true;
            }
        }
        // if the value of the index array is -1 we have checked all pairs for this particle.
        // checkme: how often does this happen?
        else {
            // end for p loop
            return false;
        }
    }
    return false;
}



__global__ void simulate(double *savedata, double *dx2, double *dx4, int *Bounds, hiprandStatePhilox4_32_10_t *state,
                         double *SimulationParams,
                         double4 *d4swc, int *nlut, int *NewIndex, int *IndexSize, int size, int iter, bool debug,
                         double3 point, int SaveAll) {
    int gid = threadIdx.x + blockDim.x * blockIdx.x;

    if (gid < size) {
        double step_size = SimulationParams[2];
        double perm_prob = SimulationParams[3];
        int init_in = (int) SimulationParams[4];
        double vsize = SimulationParams[9];
        double3 A;
        int2 parstate;

        // int3 gx = make_int3(3 * gid + 0, 3 * gid + 1, 3 * gid + 2);
        // define variables for loop
        double4 xi;
        double3 nextpos;
        double3 xnot;
        int3 upper;
        int3 lower;
        int3 floorpos;
        int3 b_int3 = make_int3(Bounds[0], Bounds[1], Bounds[2]);
        int3 i_int3 = make_int3(IndexSize[0], IndexSize[1], IndexSize[2]);
        double3 d2 = make_double3(0.0, 0.0, 0.0);
        bool completes;
        bool flag;
        double step = step_size;

        /////////////////////
        // signal variables
        {
        /*

        double s0 = 0; // Signal weighted by T2 relaxation
        double t[Nc_max] = {0}; // The time staying in compartments

        */
        }
        ////////////////////

        // init local state var
        hiprandStatePhilox4_32_10_t localstate = state[gid];
        xi = hiprand_uniform4_double(&localstate);

        // initialize position inside cell
        A = initPosition(gid, dx2, Bounds, state, SimulationParams, d4swc, nlut, NewIndex, IndexSize,
                         size, iter, init_in, debug, point);

        // record initial position
        xnot = make_double3(A.x, A.y, A.z);

        // flag is initially false
        flag = false;

        // state is based on intialization conditions if particles are required to start inside then parstate -> [1,1]
        // todo figure out how to get parstate from init position function... requires a global parstate.
        parstate = make_int2(1, 1);

        // parlut defines whether particle is within bounds of LUT
        int parlut = 1;

        // iterate over steps
        for (int i = 0; i < iter; i++) {

            if (flag == false) {
                // generate uniform randoms for step
                xi = hiprand_uniform4_double(&localstate);

                // set next position
                double theta = 2 * PI * xi.x;
                double v = xi.y;
                double cos_phi = 2 * v - 1;
                double sin_phi = sqrt(1 - pow(cos_phi, 2));
                nextpos.x = A.x + (step * sin_phi * cos(theta));
                nextpos.y = A.y + (step * sin_phi * sin(theta));
                nextpos.z = A.z + (step * cos_phi);

                // floor of next position -> check voxels
                floorpos = make_int3((int) nextpos.x, (int) nextpos.y, (int) nextpos.z);

                // upper bounds of lookup table
                upper = make_int3(floorpos.x < b_int3.x, floorpos.y < b_int3.y, floorpos.z < b_int3.z);

                // lower bounds of lookup table
                lower = make_int3(floorpos.x >= 0, floorpos.y >= 0, floorpos.z >= 0);

                // position inside the bounds of volume -> state of next position true : false
                parlut = (lower.x && lower.y && lower.z && upper.x && upper.y && upper.z) ? 1 : 0;

            

                if (parlut == 0) {
                    // do something
                    // reflection
                    int3 aob;
                    aob.x = (lower.x && upper.x) ? 0 : 1;
                    aob.y = (lower.y && upper.y) ? 0 : 1;
                    aob.z = (lower.z && upper.z) ? 0 : 1;

                    theta = 2 * PI * xi.x;
                    v = xi.y;
                    cos_phi = 2 * v - 1;
                    sin_phi = sqrt(1 - pow(cos_phi, 2));

                    if (aob.x) {
                        nextpos.x = A.x - step * sin_phi * cos(theta);
                    }

                    if (aob.y) {
                        nextpos.y = A.y - step * sin_phi * sin(theta);
                    }

                    if (aob.z) {
                        nextpos.z = A.z - step * cos_phi;
                    }

                    // floor of next position -> check voxels
                    floorpos = make_int3((int) nextpos.x, (int) nextpos.y, (int) nextpos.z);

                    // upper bounds of lookup table
                    upper = make_int3(floorpos.x < b_int3.x, floorpos.y < b_int3.y, floorpos.z < b_int3.z);

                    // lower bounds of lookup table
                    lower = make_int3(floorpos.x >= 0, floorpos.y >= 0, floorpos.z >= 0);

                    // position inside the bounds of volume -> state of next position true : false
                    parlut = (lower.x && lower.y && lower.z && upper.x && upper.y && upper.z) ? 1 : 0;

                    if (parlut == 0) {
                        printf("X: %d\tY: %dZ\t: %d\tResolved::::%d\n", aob.x, aob.y, aob.z, parstate.y);
                    }

                }

                // extract value of lookup @ index

                // if parstate.y == 0 then we failed to reflect the particle back into the volume.
                // throw an error and exit the simulation
                // if (parlut == 0) {
                //     printf("Particle %d failed to reflect back into the volume. Exiting simulation.\n", gid);
                //     exit(1);
                // }

                // reset particle state for next conditionals
                parstate.y = 0; // checkme: is this necessary or valid?

                // sub2ind
                int id_test = s2i(floorpos, b_int3);

                // extract lookup table value
                int test_lutvalue = nlut[id_test];

                // child parent indicies
                int2 vindex;

                // parent swc values
                double4 parent;

                // child swc values
                double4 child;

                // distance^2 from child to parent
                double dist2;

                // for each connection check if particle inside

                // pre function definition
                {
                  /*
                    for (int page = 0; page < i_int3.z; page++) {

                      // create a subscript indices
                      int3 c_new = make_int3(test_lutvalue, 0, page);
                      int3 p_new = make_int3(test_lutvalue, 1, page);

                      // convert subscripted index to linear index and get value from Index Array
                      vindex.x = NewIndex[s2i(c_new, i_int3)] - 1;
                      vindex.y = NewIndex[s2i(p_new, i_int3)] - 1;

                      if ((vindex.x) != -1) {
                          //extract child parent values from swc
                          child = d4swc[vindex.x];
                          parent = d4swc[vindex.y];

                          // calculate euclidean distance
                          dist2 = pow(parent.x - child.x, 2) + pow(parent.y - child.y, 2) +
                                  pow(parent.z - child.z, 2);

                          // determine whether particle is inside this connection
                          bool inside = swc2v(nextpos, child, parent, dist2);

                          // if it is inside the connection we don't need to check the remaining.
                          if (inside) {
                              // update the particles state
                              parstate.y = 1;

                              // end for p loop
                              page = i_int3.z;
                          }
                      }

                          // if the value of the index array is -1 we have checked all pairs for this particle.
                          // checkme: how often does this happen?
                      else {
                          // printf("No Cons Found: Particle %d \t Step %d\n", gid, step);
                          // end for p loop
                          page = i_int3.z;
                          parstate.y = 0;
                      }
                  }
                */
                }



                bool inside = checkConnections(i_int3, test_lutvalue, nextpos, NewIndex, d4swc);
                if (inside) {
                    // update the particles state
                    parstate.y = 1;
                } else {
                    parstate.y = 0;
                }

                // determine if step executes
                completes = xi.w < perm_prob;
                // completes.x = xi.w < perm.x;
                // completes.y = xi.w < perm.y;

                /**
                * particle inside? 0 0 - update
                * particle inside? 0 1 - check if updates
                * particle inside? 1 0 - check if updates
                * particle inside? 1 1 - update
                */

                // particle inside: [0 0] || [1 1]
                if (parstate.x == parstate.y) { A = nextpos; }

                // particle inside: [1 0]
                if (parstate.x && !parstate.y) {
                    if (completes == true) {
                        A = nextpos;
                        parstate.x = parstate.y;
                    } else {

                    }
                }

                // particle inside [0 1]
                if (!parstate.x && parstate.y) {
                    if (completes == true) {
                        A = nextpos;
                        parstate.x = parstate.y;
                    } else {

                    }
                }
            } else {
                // update flag for next step
                flag = false;
            }

            // Store Position Data
            if (SaveAll) {
                int3 dix = make_int3(size, iter, 3);
                int3 did[4];
                did[0] = make_int3(gid, i, 0); did[1] = make_int3(gid, i, 1); did[2] = make_int3(gid, i, 2);
                // did[3] = make_int3(gid, i, 3);

                did[3] = make_int3(s2i(did[0],dix),s2i(did[1],dix),s2i(did[2],dix));
                // int4 allid = make_int4(s2i(did[0], dix), s2i(did[1], dix), s2i(did[2], dix), s2i(did[3], dix));

                savedata[did[3].x] = A.x;
                savedata[did[3].y] = A.y;
                savedata[did[3].z] = A.z;
                // savedata[allid.w] = (double) flag;
            }

            // Store Tensor Data
            {
                // diffusionTensor(A, xnot, vsize, dx2, savedata, d2, i, gid, iter, size);

                // calculate displacement
                // d2.x = fabs((A.x - xnot.x) * vsize);
                // d2.y = fabs((A.y - xnot.y) * vsize);
                // d2.z = fabs((A.z - xnot.z) * vsize);

                diffusionTensor(&A, &xnot, vsize, dx2, dx4, &d2, i,  gid,  iter,  size);

                // Diffusion Tensor
                {
                    /*
                    atomicAdd(&dx2[6 * i + 0], d2.x * d2.x);
                    atomicAdd(&dx2[6 * i + 1], d2.x * d2.y);
                    atomicAdd(&dx2[6 * i + 2], d2.x * d2.z);
                    atomicAdd(&dx2[6 * i + 3], d2.y * d2.y);
                    atomicAdd(&dx2[6 * i + 4], d2.y * d2.z);
                    atomicAdd(&dx2[6 * i + 5], d2.z * d2.z);
                    */
                }

                // Kurtosis Tensor
                {
                    /*
                    atomicAdd(&dx4[15 * i + 0], d2.x * d2.x * d2.x * d2.x);
                    atomicAdd(&dx4[15 * i + 1], d2.x * d2.x * d2.x * d2.y);
                    atomicAdd(&dx4[15 * i + 2], d2.x * d2.x * d2.x * d2.z);
                    atomicAdd(&dx4[15 * i + 3], d2.x * d2.x * d2.y * d2.y);
                    atomicAdd(&dx4[15 * i + 4], d2.x * d2.x * d2.y * d2.z);
                    atomicAdd(&dx4[15 * i + 5], d2.x * d2.x * d2.z * d2.z);
                    atomicAdd(&dx4[15 * i + 6], d2.x * d2.y * d2.y * d2.y);
                    atomicAdd(&dx4[15 * i + 7], d2.x * d2.y * d2.y * d2.z);
                    atomicAdd(&dx4[15 * i + 8], d2.x * d2.y * d2.z * d2.z);
                    atomicAdd(&dx4[15 * i + 9], d2.x * d2.z * d2.z * d2.z);
                    atomicAdd(&dx4[15 * i + 10], d2.y * d2.y * d2.y * d2.y);
                    atomicAdd(&dx4[15 * i + 11], d2.y * d2.y * d2.y * d2.z);
                    atomicAdd(&dx4[15 * i + 12], d2.y * d2.y * d2.z * d2.z);
                    atomicAdd(&dx4[15 * i + 13], d2.y * d2.z * d2.z * d2.z);
                    atomicAdd(&dx4[15 * i + 14], d2.z * d2.z * d2.z * d2.z);
                    */
                }


                // Signal
                /*{

                    s0 = 0.0;
                    for (int j = 0; j < Nc; j++) {
                        s0 = s0 + (t[j] / T2[j]);
                    }

                    s0 = exp(-1.0 * s0);
                    atomAdd(&sig0[tidx], s0);

                    // loop over b values
                    for (int j = 0; j < Nbvec; j++) {
                        qx = sqrt(bval[j] / TD[tidx]) * (d2.x * bvec[j * 3 + 0] + d2.y * bvec[j * 3 + 1] + d2.z * bvec[j * 3 + 2]);
                        // qx = sqrt(bval[j] / TD[tidx]) * (dx * bvec[j * 3 + 0] + dy * bvec[j * 3 + 1] + dz * bvec[j * 3 + 2]);
                        atomAdd(&sigRe[Nbvec * tidx + j], s0 * cos(qx));
                    }
                }*/
            }

        }
    }
}

int main(int argc, char *argv[]) {
    hipEvent_t start_c, stop_c;
    hipEventCreate(&start_c);
    hipEventCreate(&stop_c);
    float milliseconds = 0;
    system("clear");
    int size = 10;
    int iter = 10;
    int SaveAll;
    std::string path;
    controller control;

    /**
     * Read Simulation and Initialize Object
     */
    // Parse Arguments
    if (argc < 2) {
      // "/autofs/space/symphony_002/users/BenSylvanus/cuda/Sims/data"
        path = "/autofs/space/symphony_002/users/BenSylvanus/cuda/Sims";
        std::string InPath = path;
        std::string OutPath = path;

        InPath.append("/data"); OutPath.append("/results");
        control.Setup(InPath,OutPath,0);
        control.start();
    }
    else {

      control.Setup(argc, argv, 1);
    }

    system("clear");

    double simparam[10];
    simulation sim = control.getSim();
    printf("Path: %s\n",sim.getResultPath().c_str());
    path = sim.getResultPath();
    size = sim.getParticle_num();
    iter = sim.getStep_num();
    std::vector<double> simulationparams = sim.getParameterdata();
    SaveAll = sim.getSaveAll();

    if (SaveAll)
    {
      printf("Executed True\n");

    }
    else {
      printf("Executed False\n");
    }


    for (int i = 0; i < 10; i++) {
        double value = simulationparams[i];
        simparam[i] = value;
    }
    int block_size = 256;
    dim3 block(block_size);
    dim3 grid((size / block.x) + 1);

    std::vector <uint64_t> bounds = sim.getbounds();
    int boundx = (int) bounds[0];
    int boundy = (int) bounds[1];
    int boundz = (int) bounds[2];
    int prod = (int) (boundx * boundy * boundz);
    std::vector<double> r_swc = sim.getSwc();
    int nrow = r_swc.size() / 6;

    //old comments
    {
        /** 
         * @brief Simulation Params Array
         * <li> particle_num = SimulationParams[0] </li>
         * <li> step_num = SimulationParams[1] </li>
         * <li> step_size = SimulationParams[2] </li>
         * <li> perm_prob = SimulationParams[3] </li>
         * <li> init_in = SimulationParams[4] </li>
         * <li> D0 = SimulationParams[5] </li>
         * <li> d = SimulationParams[6] </li>
         * <li> scale = SimulationParams[7] </li>
         * <li> tstep = SimulationParams[8] </li>
         * <li> vsize = SimulationParams[9] </li>
         * @brief INDEXING SWC ARRAY
         * index + (dx*0:5);
         * row+(nrow*col)
         * Example: nrow = 10; get all elements from row 1;
         * swc[1,0:5] ---->
         * <li> swc(1,0) = swc[1+10*0];</li>
         * <li> swc(1,1) = swc[1+10*1];</li>
         * <li> swc(1,2) = swc[1+10*2];</li>
         * <li> swc(1,3) = swc[1+10*3];</li>
         * <li> swc(1,4) = swc[1+10*4];</li>
         * <li> swc(1,5) = swc[1+10*5];</li>
         */
        //we only need the x y z r    float milliseconds = 0; of our swc array.
        // stride + bx * (by * y + z) + x
        // int id0 = 0 + (boundx) * ((boundy) * 2 + 2) + 3;
        // printf("lut[%d]: %d\n", id0, lut[id0]);
        // ----------------------
        // Lookup Table Summary
        // linearindex = stride + bx * (by * z + y) + x
        // voxel coord: (x,y,z);
        // ----------------------
    }

    double4 swc_trim[nrow];
    double w_swc[nrow * 4];
    for (int i = 0; i < nrow; i++) {
        swc_trim[i].x = r_swc[i + nrow * 1];
        swc_trim[i].y = r_swc[i + nrow * 2];
        swc_trim[i].z = r_swc[i + nrow * 3];
        swc_trim[i].w = r_swc[i + nrow * 4];
    }
    for (int i = 0; i < nrow; i++) {
        w_swc[4 * i + 0] = r_swc[i + nrow * 1];
        w_swc[4 * i + 1] = r_swc[i + nrow * 2];
        w_swc[4 * i + 2] = r_swc[i + nrow * 3];
        w_swc[4 * i + 3] = r_swc[i + nrow * 4];
    }

    std::vector <uint64_t> lut = sim.getLut();
    std::vector <uint64_t> indexarr = sim.getIndex();
    std::vector <std::vector<uint64_t>> arrdims = sim.getArraydims();
    std::vector <uint64_t> swc_dims = arrdims[0];
    std::vector <uint64_t> lut_dims = arrdims[1];
    std::vector <uint64_t> index_dims = arrdims[2];
    std::vector <uint64_t> pairs_dims = arrdims[3];
    std::vector <uint64_t> bounds_dims = arrdims[4];
    int newindexsize = index_dims[0] * index_dims[1] * index_dims[2];

    /**
     * Host Section:
     * - Create Pointers
     * - Allocate Memory
     * - Set Values
     */
    // Create Host Pointers
    int *hostBounds;
    double *hostdx2;
    double *hostdx4;
    double *hostSimP;
    int *hostNewLut;
    int *hostNewIndex;
    int *hostIndexSize;
    double4 *hostD4Swc;
    double *mdx2;
    double *mdx4;
    double *hostAllData;

    // Alloc Memory for Host Pointers
    {

        hostBounds = (int *) malloc(3 * sizeof(int));
        hostdx2 = (double *) malloc(6 * iter * sizeof(double));
        hostdx4 = (double *) malloc(15 * iter * sizeof(double));
        hostSimP = (double *) malloc(10 * sizeof(double));
        hostD4Swc = (double4 *) malloc(nrow * sizeof(double4));
        hostNewLut = (int *) malloc(prod * sizeof(int));
        hostNewIndex = (int *) malloc(newindexsize * sizeof(int));
        hostIndexSize = (int *) malloc(3 * sizeof(int));
        mdx2 = (double *) malloc(6 * iter * sizeof(double));
        mdx4 = (double *) malloc(15 * iter * sizeof(double));
        if (SaveAll)
        {
          hostAllData = (double *) malloc(3 * iter * size * sizeof(double));
        }
        else {
          hostAllData = (double *) malloc(3 * sizeof(double));
        }

        printf("Allocated Host Data\n");
    }

    // Set Values for Host
    {

        hostBounds[0] = boundx;
        hostBounds[1] = boundy;
        hostBounds[2] = boundz;
        memset(hostdx2, 0.0, 6 * iter * sizeof(double));
        memset(hostdx4, 0.0, 15 * iter * sizeof(double));
        {

            for (int i = 0; i < 10; i++) {
                hostSimP[i] = simparam[i];
            }

            for (int i = 0; i < nrow; i++) {
                hostD4Swc[i].x = swc_trim[i].x;
                hostD4Swc[i].y = swc_trim[i].y;
                hostD4Swc[i].z = swc_trim[i].z;
                hostD4Swc[i].w = swc_trim[i].w;
            }

            for (int i = 0; i < prod; i++) {
                int value = lut[i];
                hostNewLut[i] = value;
            }

            for (int i = 0; i < indexarr.size(); i++) {
                int value = indexarr[i];
                hostNewIndex[i] = value;
            }

            for (int i = 0; i < 3; i++) {
                int value = index_dims[i];
                hostIndexSize[i] = value;
            }

        }
        memset(mdx2, 0.0, 6 * iter * sizeof(double));
        memset(mdx4, 0.0, 15 * iter * sizeof(double));

        if (SaveAll) {
          memset(hostAllData, 0.0, 3 * iter * size * sizeof(double));
        }
        else {
          memset(hostAllData, 0.0, 3 * sizeof(double));
        }

        printf("Set Host Values\n");
    }

    /**
     * Device Section:
     * - Create Pointers
     * - Allocate Memory
     * - Set Values
     */
    // Create Device Pointers
    hiprandStatePhilox4_32_10_t *deviceState;
    double *devicedx2;
    double *devicedx4;
    int *deviceBounds;
    double *deviceSimP;
    double4 *deviced4Swc;
    int *deviceNewLut;
    int *deviceNewIndex;
    int *deviceIndexSize;
    double *deviceAllData;

    clock_t start = clock();
    hipEventRecord(start_c);

    // Allocate Memory on Device
    {

        hipMalloc((double **) &devicedx2, 6 * iter * sizeof(double));
        hipMalloc((double **) &devicedx4, 15 * iter * sizeof(double));
        hipMalloc((int **) &deviceBounds, 3 * sizeof(int));
        hipMalloc((hiprandStatePhilox4_32_10_t * *) & deviceState, size * sizeof(hiprandStatePhilox4_32_10_t));
        hipMalloc((double **) &deviceSimP, 10 * sizeof(double));
        hipMalloc((double4 * *) & deviced4Swc, nrow * sizeof(double4));
        hipMalloc((int **) &deviceNewLut, prod * sizeof(int));
        hipMalloc((int **) &deviceNewIndex, newindexsize * sizeof(int));
        hipMalloc((int **) &deviceIndexSize, 3 * sizeof(int));
        if (SaveAll)
        {
          hipMalloc((double **) &deviceAllData, 3 * iter * size * sizeof(double));
        }
        else {
          hipMalloc((double **) &deviceAllData, 3 * sizeof(double));
        }
        printf("Device Memory Allocated\n");
    }

    // Set Values for Device
    {
        printf("Copying Host data to Device\n");
        hipMemcpy(devicedx2, hostdx2, 6 * iter * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(devicedx4, hostdx4, 15 * iter * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(deviceBounds, hostBounds, 3 * sizeof(int), hipMemcpyHostToDevice);
        setup_kernel<<<grid, block>>>(deviceState, 1);
        hipMemcpy(deviceSimP, hostSimP, 10 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(deviced4Swc, hostD4Swc, nrow * sizeof(double4), hipMemcpyHostToDevice);
        hipMemcpy(deviceNewLut, hostNewLut, prod * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(deviceNewIndex, hostNewIndex, newindexsize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(deviceIndexSize, hostIndexSize, 3 * sizeof(int), hipMemcpyHostToDevice);
        if (SaveAll)
        {
          hipMemcpy(deviceAllData, hostAllData, 3 * iter * size * sizeof(double), hipMemcpyHostToDevice);
        }
        else
        {
          hipMemcpy(deviceAllData, hostAllData, 3 * sizeof(double), hipMemcpyHostToDevice);
        }
    }

    // option for printing in kernel
    bool debug = false;
    double3 point = make_double3(hostD4Swc[0].x, hostD4Swc[0].y, hostD4Swc[0].z);

    /**
     * Call Kernel
    */
    printf("Simulating...\n");

    // kernel
    {

        simulate<<<grid, block>>>(deviceAllData, devicedx2, devicedx4, deviceBounds, deviceState, deviceSimP,
                                  deviced4Swc,
                                  deviceNewLut, deviceNewIndex, deviceIndexSize, size, iter, debug, point, SaveAll);
        hipEventRecord(stop_c);
    }

    // Wait for results
    hipDeviceSynchronize();

    clock_t end = clock();
    double gpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Simulation took %f seconds\n", gpu_time_used);

    /**
     * Copy Results From Device to Host
     */
    printf("Copying back to Host\n");

    // hipMemcpyDeviceToHost
    {

        hipMemcpy(hostdx2, devicedx2, 6 * iter * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(hostdx4, devicedx4, 15 * iter * sizeof(double), hipMemcpyDeviceToHost);
        if (SaveAll)
        {
            hipMemcpy(hostAllData, deviceAllData, 3 * iter * size * sizeof(double), hipMemcpyDeviceToHost);
        }
        else {
            hipMemcpy(hostAllData, deviceAllData, 3 * sizeof(double), hipMemcpyDeviceToHost);
        }
        // hipMemcpy(hostInitPos, deviceInitPos, 3 * size * sizeof(double), hipMemcpyDeviceToHost);
    }

    hipEventSynchronize(stop_c);
    hipEventElapsedTime(&milliseconds, start_c, stop_c);
    end = clock();
    printf("Kernel took %f seconds\n", milliseconds / 1e3);
    auto t1 = high_resolution_clock::now();

    // Free Device Memory
    {

        printf("Freeing Device Data: ");
        hipFree(deviceBounds);
        hipFree(deviceState);
        hipFree(devicedx2);
        hipFree(devicedx4);
        hipFree(deviceSimP);
        hipFree(deviced4Swc);
        hipFree(deviceNewIndex);
        hipFree(deviceIndexSize);
        hipFree(deviceAllData);
    }

    auto t2 = high_resolution_clock::now();
    duration<double, std::milli> ms_double = t2 - t1;
    printf("%f seconds\n", ms_double.count() / 1e3);
    printf("Writing results: ");

    // Write Results
    {

        std::string outpath = sim.getResultPath();
        t1 = high_resolution_clock::now();
        writeResults(hostdx2, hostdx4, mdx2, mdx4, hostSimP, w_swc, iter, size, nrow, outpath);
        std::string allDataPath = outpath;
        if (SaveAll)
        {
        allDataPath.append("/allData.bin");
        FILE *outFile = fopen(allDataPath.c_str(), "wb");
        fwrite(hostAllData, sizeof(double), iter * size * 3, outFile);
        fclose(outFile);
      }
    }

    t2 = high_resolution_clock::now();
    ms_double = t2 - t1;
    printf("%f seconds\n", ms_double.count() / 1e3);

    // Free Host Memory
    {

        free(hostBounds);
        free(hostdx2);
        free(hostdx4);
        free(hostSimP);
        free(hostD4Swc);
        free(hostNewIndex);
        free(hostIndexSize);
        free(mdx2);
        free(mdx4);
        free(hostAllData);
    }

    printf("Done!\n");
    return 0;
}
