#include "hip/hip_runtime.h"
#include "./src/simreader.h"
#include "./src/simulation.h"
#include "./src/controller.h"
#include "./src/viewer.h"
#include "./src/funcs.h"
#include "./src/overloads.h"
#include "vector"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <cstring>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <chrono>
#include <thread>
#include <hipblas.h>

#define HIP_PI_F 3.141592654f
#define PI 3.14159265358979323846
#define LDPI 3.141592653589793238462643383279502884L
#define timepoints 1000
#define Nc 2
#define Nbvec 3
#define SOD (sizeof(double))
#define SOI (sizeof(int))
#define SOF (sizeof(float))
#define SOD3 (sizeof(double3))
#define SOD4 (sizeof(double4))
#define SOI3 (sizeof(int3))
#define SOI4 (sizeof(int4))

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

using std::cout;
using std::cin;
using std::endl;
using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


/**
 * @brief Initializes the random number generator
 * @param state pointer to the random number generator
 * @param seed seed for the random number generator
 */
__global__ void setup_kernel(hiprandStatePhilox4_32_10_t *state, unsigned long seed) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}


/**
 * @brief Simulation Kernel for the GPU
 * @param savedata - the data to be saved
 * @param dx2 - the second moment of the diffusion tensor
 * @param dx4 - the fourth moment of the diffusion tensor
 * @param Bounds - the bounds of the simulation
 * @param state - the random number generator state
 * @param SimulationParams - the simulation parameters
 * @param d4swc - the swc data
 * @param nlut - the neighbor lookup table
 * @param NewIndex - the new index
 * @param IndexSize - the index size
 * @param size - the number of particles
 * @param iter - the number of iterations
 * @param debug - whether or not to print debug statements
 * @param point - the point to simulate
 * @param SaveAll - whether or not to save all data
 * @param Reflections - the reflections
 * @param Uref - the unreflected data
 * @param flip - the flip data
 * @param T2 - the T2 data
 * @param T - the T data
 * @param Sig0 - the Sig0 data
 * @param SigRe - the SigRe data
 * @param BVec - the BVec data
 * @param BVal - the BVal data
 * @param TD - the TD data
 */
__global__ void simulate(double *savedata, double *dx2, double *dx4, int3 Bounds, hiprandStatePhilox4_32_10_t *state,
                         double *SimulationParams,
                         double4 *d4swc, int *nlut, int *NewIndex, int3 IndexSize, int size, int iter, bool debug,
                         double3 point, int SaveAll, double * Reflections, double * Uref, int * flip,
                         double * T2, double * T, double * Sig0, double * SigRe, double* BVec, double * BVal, double * TD) {

    int gid = threadIdx.x + blockDim.x * blockIdx.x;
    if (gid < size) {
        /**
            @index particle_num = SimulationParams[0]
            @index step_num = SimulationParams[1]
            @index step_size = SimulationParams[2]
            @index perm_prob = SimulationParams[3]
            @index init_in = SimulationParams[4]
            @index D0 = SimulationParams[5]
            @index d = SimulationParams[6]
            @index scale = SimulationParams[7]
            @index tstep = SimulationParams[8]
        */
        double step_size = SimulationParams[2];
        double perm_prob = SimulationParams[3];
        int init_in = (int) SimulationParams[4];
        double tstep = SimulationParams[8];
        double vsize = SimulationParams[9];
        double3 A;
        int2 parstate;
        double4 xi;
        double3 nextpos;
        double3 xnot;
        int3 upper;
        int3 lower;
        int3 floorpos;
        int Tstep=iter/timepoints;
        double fstep = 1;

        int3 b_int3 = make_int3(Bounds.x, Bounds.y, Bounds.z);
        int3 i_int3 = make_int3(IndexSize.x, IndexSize.y, IndexSize.z);
    
        double _T2[Nc]; 
        double t2[3] = {80, 40, 60};
        for (int j = 0; j < Nc; j++)
        {
            _T2[j] = t2[j];
        }

        double3 d2 = make_double3(0.0, 0.0, 0.0);
        bool completes;
        bool flag;
        double step = step_size;

        // init local state var
        hiprandStatePhilox4_32_10_t localstate = state[gid];
        xi = hiprand_uniform4_double(&localstate);

        // initialize position inside cell
        A = initPosition(gid, dx2, Bounds, state, SimulationParams, d4swc, nlut, NewIndex, IndexSize,
                         size, iter, init_in, debug, point);

        // record initial position
        xnot = make_double3(A.x, A.y, A.z);

        // flag is initially false
        flag = false;

        // state is based on intialization conditions if particles are required to start inside then parstate -> [1,1]
        parstate = make_int2(1, 1);

        // parlut defines whether particle is within bounds of LUT
        int parlut = 1;
        double t[Nc] ={0}; // add tstep for step in compartment

        // iterate over steps
        for (int i = 0; i < iter; i++) {
                // generate uniform randoms for step
                xi = hiprand_uniform4_double(&localstate);

                // determine if step executes
                completes = xi.w < perm_prob;

                // compute next position
                double pi = PI;
                computeNext(A, step, xi, nextpos, pi);

                // check coordinate validity
                validCoord(nextpos, A, b_int3, upper, lower, floorpos, Reflections, Uref, gid, i, size, iter, flip);

                // floor of next position -> check voxels
                floorpos = make_int3((int) nextpos.x, (int) nextpos.y, (int) nextpos.z);

                // extract lookup table value
                int test_lutvalue = nlut[s2i(floorpos,b_int3)];

                // for each connection check if particle inside
                bool inside = checkConnections(i_int3, test_lutvalue, nextpos, NewIndex, d4swc, fstep);

                parstate.y = (inside) ? 1 : 0;

                /**
                * @cases particle inside? 0 0 - update
                * @cases particle inside? 0 1 - check if updates
                * @cases particle inside? 1 0 - check if updates
                * @cases particle inside? 1 1 - update
                */

                // particle inside: [0 0] || [1 1]
                if (parstate.x == parstate.y) { 
                    A = nextpos; 
                    if (parstate.x) {
                        t[0] = t[0] + tstep;
                    } else {
                        t[1] = t[1] + tstep;
                    }
                }

                // particle inside: [1 0]
                if (parstate.x && !parstate.y) {
                    if (completes == true) {
                        A = nextpos;
                        parstate.x = parstate.y;
                        t[0] = t[0] + tstep * fstep;
                        t[1] = t[1] + tstep * (1 - fstep);
                    } else {
                        t[0] = t[0] + tstep;
                    }
                }

                // particle inside [0 1]
                if (!parstate.x && parstate.y) {
                    if (completes == true) {
                        A = nextpos;
                        parstate.x = parstate.y;
                        t[0] = t[0] + tstep * 1-fstep;
                        t[1] = t[1] + tstep * fstep;
                    } else {
                        t[1] = t[1] + tstep;
                    }
                }

            // Store Position Data
            if (SaveAll) {
                int3 dix = make_int3(size, iter, 3);
                int3 did[4];
                did[0] = make_int3(gid, i, 0);
                did[1] = make_int3(gid, i, 1);
                did[2] = make_int3(gid, i, 2);
                did[3] = make_int3(s2i(did[0], dix), s2i(did[1], dix), s2i(did[2], dix));
                set(savedata, did[3], A);
            }
            // Store Tensor Data
            {
                diffusionTensor(&A, &xnot, vsize, dx2, dx4, &d2, i, gid, iter, size);
                // https://github.com/NYU-DiffusionMRI/monte-carlo-simulation-3D-RMS/blob/master/part1_demo3_simulation.m

            }

            // Store Signal Data
            {

                if (i%Tstep == 0)
                {
                    int tidx=i/Tstep;
                    // loop over compartments
                    double s0 = 0.0;
                    for (int j = 0; j < 2; j++) {
                        /**
                            * @var s0 is our summation variable
                            * @var t[j] is the time in compartment j
                            * @var T2 is the T2 Relaxation in Compartment j
                        */  
                            s0 = s0 + (double) (t[j] / _T2[j]); // TODO implement "t" as time in each compartment

                    }


                    s0 = exp(-1.0 * s0);
                    atomicAdd(&Sig0[tidx],s0);
                    for (int j = 0; j < Nc; j++)
                    {
                        t[j]= 0;
                    }
                }
                // Signal
        

                /*{
                    // loop over b values
                    for (int j = 0; j < Nbvec; j++) {
                        // bval is the b value
                        // bvec is the gradient direction vector
                        // TD is the diffusion time
                        // qx is the q value
                        // d2.x, d2.y, d2.z are the displacements defined as fabs((A - xnot) * vsize);
                        // get the bvec for the current b value
                        bd = make_double3(bvec[j * 3 + 0], bvec[j * 3 + 1], bvec[j * 3 + 2]);
                        bv = bval[j];
                        td = TD[tidx];
                        // qx = sqrt(bv / td) * dot(d2, bd);
                        qx = sqrt(bv / td) * (d2.x * bd.x + d2.y * bd.y + d2.z * bd.z);
                        qx = sqrt(bval[j] / TD[tidx]) * (d2.x * bvec[j * 3 + 0] + d2.y * bvec[j * 3 + 1] + d2.z * bvec[j * 3 + 2]);
                        // qx = sqrt(bval[j] / TD[tidx]) * (dx * bvec[j * 3 + 0] + dy * bvec[j * 3 + 1] + dz * bvec[j * 3 + 2]);
                        atomAdd(&sigRe[Nbvec * tidx + j], s0 * cos(qx));
                    }
                }*/
            }
        }
    }
}

int main(int argc, char *argv[]) {
    hipEvent_t start_c, stop_c;
    hipEventCreate(&start_c);
    hipEventCreate(&stop_c);
    float milliseconds = 0;
    system("clear");
    int size = 10;
    int iter = 10;
    int SaveAll;
    std::string path;
    controller control;

    /**
     * Read Simulation and Initialize Object
     */
    // Parse Arguments
    if (argc < 2) {
        path = "/autofs/space/symphony_002/users/BenSylvanus/cuda/Sims";
        std::string InPath = path;
        std::string OutPath = path;
        InPath.append("/data");
        OutPath.append("/results");
        control.Setup(InPath, OutPath, 0);
        control.start();
    } else {
        control.Setup(argc, argv, 1);
    }

    system("clear");

    double simparam[10];
    simulation sim = control.getSim();
    printf("Path: %s\n", sim.getResultPath().c_str());
    path = sim.getResultPath();
    size = sim.getParticle_num();
    iter = sim.getStep_num();
    std::vector<double> simulationparams = sim.getParameterdata();
    SaveAll = sim.getSaveAll();

    for (int i = 0; i < 10; i++) {
        double value = simulationparams[i];
        simparam[i] = value;
    }

    int block_size = 256;
    dim3 block(block_size);
    dim3 grid((size / block.x) + 1);

    std::vector <uint64_t> bounds = sim.getbounds();
    int boundx = (int) bounds[0]; int boundy = (int) bounds[1]; int boundz = (int) bounds[2];
    int prod = (int) (boundx * boundy * boundz);
    std::vector<double> r_swc = sim.getSwc();
    int nrow = r_swc.size() / 6;

    double4 swc_trim[nrow];
    double w_swc[nrow * 4];
    for (int i = 0; i < nrow; i++) {
        swc_trim[i].x = r_swc[i + nrow * 1];
        swc_trim[i].y = r_swc[i + nrow * 2];
        swc_trim[i].z = r_swc[i + nrow * 3];
        swc_trim[i].w = r_swc[i + nrow * 4];
    }
    for (int i = 0; i < nrow; i++) {
        w_swc[4 * i + 0] = r_swc[i + nrow * 1];
        w_swc[4 * i + 1] = r_swc[i + nrow * 2];
        w_swc[4 * i + 2] = r_swc[i + nrow * 3];
        w_swc[4 * i + 3] = r_swc[i + nrow * 4];
    }

    std::vector <uint64_t> lut = sim.getLut();
    std::vector <uint64_t> indexarr = sim.getIndex();
    std::vector <std::vector<uint64_t>> arrdims = sim.getArraydims();
    std::vector <uint64_t> swc_dims = arrdims[0];
    std::vector <uint64_t> lut_dims = arrdims[1];
    std::vector <uint64_t> index_dims = arrdims[2];
    std::vector <uint64_t> pairs_dims = arrdims[3];
    std::vector <uint64_t> bounds_dims = arrdims[4];
    int newindexsize = index_dims[0] * index_dims[1] * index_dims[2];

    ///Host Section

    // Create Host Pointers
    double *hostdx2;
    double *hostdx4;
    double *hostSimP;
    int *hostNewLut;
    int *hostNewIndex;
    double4 *hostD4Swc;
    double *mdx2;
    double *mdx4;
    double *hostAllData;
    double *hostReflections;
    double *hosturef;
    int* hostFlip;
    double *hostT2; // Nc * 1
    double *hostT; // Nc * 1
    double *hostSigRe; // Nbvec * iter
    double *hostSig0; // Nc * iter
    double *hostbvec; // Nbvec * 3 (x,y,z)
    double *hostbval; // Nbvec * 1 (b)
    double *hostTD;   // TD is the time elapsed at timepoint i.

    // Alloc Memory for Host Pointers
    {

        hostdx2 = (double *) malloc(6 * iter * SOD);
        hostdx4 = (double *) malloc(15 * iter * SOD);
        hostSimP = (double *) malloc(10 * SOD);
        hostD4Swc = (double4 *) malloc(nrow * SOD4);
        hostNewLut = (int *) malloc(prod * SOI);
        hostNewIndex = (int *) malloc(newindexsize * SOI);
        mdx2 = (double *) malloc(6 * iter * SOD);
        mdx4 = (double *) malloc(15 * iter * SOD);
        if (SaveAll) {
            hostAllData = (double *) malloc(3 * iter * size * SOD);
        } else {
            hostAllData = (double *) malloc(3 * SOD);
        }
        hostReflections = (double *) malloc(3 *iter * size * SOD);
        hosturef = (double *) malloc(3 *iter * size * SOD);
        hostFlip = (int *) malloc(3 * size * SOI);

        // Signal Variables
        hostT2 = (double *) malloc(Nc * SOD);
        hostT = (double *) malloc(Nc * SOD);
        hostSigRe = (double *) malloc(Nbvec * timepoints * SOD);
        hostSig0 = (double *) malloc(timepoints * SOD);
        hostbvec = (double *) malloc(Nbvec * 3 * SOD);
        hostbval = (double *) malloc(Nbvec * SOD);
        hostTD = (double *) malloc(Nbvec * SOD);
        printf("Allocated Host Data\n");
    }

    // Set Values for Host
    {
        memset(hostdx2, 0.0, 6 * iter * SOD);
        memset(hostdx4, 0.0, 15 * iter * SOD);
        {

            for (int i = 0; i < 10; i++) {
                hostSimP[i] = simparam[i];
            }

            for (int i = 0; i < nrow; i++) {
                hostD4Swc[i].x = swc_trim[i].x;
                hostD4Swc[i].y = swc_trim[i].y;
                hostD4Swc[i].z = swc_trim[i].z;
                hostD4Swc[i].w = swc_trim[i].w;
            }

            for (int i = 0; i < prod; i++) {
                int value = lut[i];
                hostNewLut[i] = value;
            }

            for (int i = 0; i < indexarr.size(); i++) {
                int value = indexarr[i];
                hostNewIndex[i] = value;
            }
        }
        memset(mdx2, 0.0, 6 * iter * SOD);
        memset(mdx4, 0.0, 15 * iter * SOD);

        if (SaveAll) {
            memset(hostAllData, 0.0, 3 * iter * size * SOD);
        } else {
            memset(hostAllData, 0.0, 3 * SOD);
        }
        memset(hostReflections, 0.0, 3 * iter * size * SOD);
        memset(hosturef, 0.0, 3 * iter * size * SOD);
        memset(hostFlip, 0.0, 3 * size * SOI);

        // signal variables
        memset(hostT2, 0.0, Nc * SOD); // T2 is read from file?
        memset(hostT, 0.0, Nc * SOD); // T is set to 0.0
        memset(hostSigRe, 0.0, Nbvec * timepoints * SOD); // Calculated in kernel
        memset(hostSig0, 0.0, timepoints * SOD); // Calculated in kernel
        memset(hostbvec, 0.0, Nbvec * 3 * SOD); // bvec is read from file
        memset(hostbval, 0.0, Nbvec * SOD); // bval is read from file
        memset(hostTD, 0.0, Nbvec * SOD); // TD is read from file
        printf("Set Host Values\n");
    }

    /**
     * Device Section:
     * - Create Pointers
     * - Allocate Memory
     * - Set Values
     */
    // Create Device Pointers
    hiprandStatePhilox4_32_10_t *deviceState;
    double *devicedx2;
    double *devicedx4;
    double *deviceSimP;
    double4 *deviced4Swc;
    int *deviceNewLut;
    int *deviceNewIndex;
    double *deviceAllData;
    double *deviceReflections;
    double *deviceURef;
    int *deviceFlip;

    // signal variables
    double *deviceT2;
    double *deviceT;
    double *deviceSigRe;
    double *deviceSig0;
    double *devicebvec;
    double *devicebval;
    double *deviceTD;


    clock_t start = clock();
    hipEventRecord(start_c);

    // Allocate Memory on Device
    {

        gpuErrchk(hipMalloc((double **) &devicedx2, 6 * iter * SOD));
        gpuErrchk(hipMalloc((double **) &devicedx4, 15 * iter * SOD));
        gpuErrchk(hipMalloc((hiprandStatePhilox4_32_10_t * *) & deviceState, size * sizeof(hiprandStatePhilox4_32_10_t)));
        gpuErrchk(hipMalloc((double **) &deviceSimP, 10 * SOD));
        gpuErrchk(hipMalloc((double4 * *) & deviced4Swc, nrow * SOD4));
        gpuErrchk(hipMalloc((int **) &deviceNewLut, prod * SOI));
        gpuErrchk(hipMalloc((int **) &deviceNewIndex, newindexsize * SOI));
        if (SaveAll) {
            gpuErrchk(hipMalloc((double **) &deviceAllData, 3 * iter * size * SOD));
        } else {
            gpuErrchk(hipMalloc((double **) &deviceAllData, 3 * SOD));
        }
        gpuErrchk(hipMalloc((double **) &deviceReflections, 3 * iter * size * SOD));
        gpuErrchk(hipMalloc((double **) &deviceURef, 3 * iter * size * SOD));
        gpuErrchk(hipMalloc((int **) &deviceFlip, 3 * size * SOI));

        // signal variables
        gpuErrchk(hipMalloc((double **) &deviceT2, Nc * SOD));
        gpuErrchk(hipMalloc((double **) &deviceT, Nc * SOD));
        gpuErrchk(hipMalloc((double **) &deviceSigRe, Nbvec * timepoints * SOD));
        gpuErrchk(hipMalloc((double **) &deviceSig0, timepoints * SOD));
        gpuErrchk(hipMalloc((double **) &devicebvec, Nbvec * 3 * SOD));
        gpuErrchk(hipMalloc((double **) &devicebval, Nbvec * SOD));
        gpuErrchk(hipMalloc((double **) &deviceTD, Nbvec * SOD));
        printf("Device Memory Allocated\n");
    }

    // Set Values for Device
    {
        printf("Copying Host data to Device\n");
        gpuErrchk(hipMemcpy(devicedx2, hostdx2, 6 * iter * SOD, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(devicedx4, hostdx4, 15 * iter * SOD, hipMemcpyHostToDevice));
        setup_kernel<<<grid, block>>>(deviceState, 1); // initialize the random states
        gpuErrchk(hipMemcpy(deviceSimP, hostSimP, 10 * SOD, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(deviced4Swc, hostD4Swc, nrow * SOD4, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(deviceNewLut, hostNewLut, prod * SOI, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(deviceNewIndex, hostNewIndex, newindexsize * SOI, hipMemcpyHostToDevice));
        if (SaveAll) {
            gpuErrchk(hipMemcpy(deviceAllData, hostAllData, 3 * iter * size * SOD, hipMemcpyHostToDevice));
        } else {
            gpuErrchk(hipMemcpy(deviceAllData, hostAllData, 3 * SOD, hipMemcpyHostToDevice));
        }
        gpuErrchk(hipMemcpy(deviceReflections, hostReflections, 3 * iter * size * SOD, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(deviceURef, hosturef, 3 * iter * size * SOD, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(deviceFlip, hostFlip, 3 * size * SOI, hipMemcpyHostToDevice));

        // signal variables
        gpuErrchk(hipMemcpy(deviceT2, hostT2, Nc * SOD, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(deviceT, hostT, Nc * SOD, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(deviceSigRe, hostSigRe, Nbvec * timepoints * SOD, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(deviceSig0, hostSig0, timepoints * SOD, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(devicebvec, hostbvec, Nbvec * 3 * SOD, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(devicebval, hostbval, Nbvec * SOD, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(deviceTD, hostTD, Nbvec * SOD, hipMemcpyHostToDevice));

    }

    // option for printing in kernel
    bool debug = false;
    double3 point = make_double3(hostD4Swc[0].x, hostD4Swc[0].y, hostD4Swc[0].z);
    int3 deviceBounds = make_int3(boundx, boundy, boundz);
    int3 deviceIndexSize = make_int3(index_dims[0], index_dims[1], index_dims[2]);



    /**
     * Call Kernel
    */


    // kernel
    {
        printf("Simulating...\n");
        simulate<<<grid, block>>>(deviceAllData, devicedx2, devicedx4, deviceBounds, deviceState, deviceSimP,
                                  deviced4Swc, deviceNewLut, deviceNewIndex, deviceIndexSize, size, iter, debug, point,
                                  SaveAll,
                                  deviceReflections, deviceURef, deviceFlip, // reflection variables
                                  deviceT2, deviceT, deviceSig0, deviceSigRe, devicebvec, devicebval, deviceTD); // signal variables
        hipEventRecord(stop_c);
    }

    // Wait for results
    hipDeviceSynchronize();

    clock_t end = clock();
    double gpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Simulation took %f seconds\n", gpu_time_used);

    /**
     * Copy Results From Device to Host
     */
    printf("Copying back to Host\n");

    // hipMemcpyDeviceToHost
    {

        gpuErrchk(hipMemcpy(hostdx2, devicedx2, 6 * iter * SOD, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(hostdx4, devicedx4, 15 * iter * SOD, hipMemcpyDeviceToHost));
        if (SaveAll) {
            gpuErrchk(hipMemcpy(hostAllData, deviceAllData, 3 * iter * size * SOD, hipMemcpyDeviceToHost));
        } else {
            gpuErrchk(hipMemcpy(hostAllData, deviceAllData, 3 * SOD, hipMemcpyDeviceToHost));
        }
        // Reflection Variables
        gpuErrchk(hipMemcpy(hostReflections, deviceReflections, 3 * iter * size * SOD, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(hosturef, deviceURef, 3 * iter * size * SOD, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(hostFlip, deviceFlip, 3 * size * SOI, hipMemcpyDeviceToHost));

        // Signal Variables
        gpuErrchk(hipMemcpy(hostT2, deviceT2, Nc * SOD, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(hostT, deviceT, Nc * SOD, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(hostSigRe, deviceSigRe, Nbvec * timepoints * SOD, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(hostSig0, deviceSig0, timepoints * SOD, hipMemcpyDeviceToHost));
    }

    hipEventSynchronize(stop_c);
    hipEventElapsedTime(&milliseconds, start_c, stop_c);
    end = clock();
    printf("Kernel took %f seconds\n", milliseconds / 1e3);
    auto t1 = high_resolution_clock::now();

    // Free Device Memory
    {

        printf("Freeing Device Data: ");
        gpuErrchk(hipFree(deviceState));
        gpuErrchk(hipFree(devicedx2));
        gpuErrchk(hipFree(devicedx4));
        gpuErrchk(hipFree(deviceSimP));
        gpuErrchk(hipFree(deviced4Swc));
        gpuErrchk(hipFree(deviceNewLut));
        gpuErrchk(hipFree(deviceNewIndex));
        gpuErrchk(hipFree(deviceAllData));

        // Reflection Variables
        gpuErrchk(hipFree(deviceReflections));
        gpuErrchk(hipFree(deviceURef));
        gpuErrchk(hipFree(deviceFlip));

        // Signal Variables
        gpuErrchk(hipFree(deviceT2));
        gpuErrchk(hipFree(deviceT));
        gpuErrchk(hipFree(deviceSigRe));
        gpuErrchk(hipFree(deviceSig0));
        gpuErrchk(hipFree(devicebvec));
        gpuErrchk(hipFree(devicebval));
        gpuErrchk(hipFree(deviceTD));

    }

    auto t2 = high_resolution_clock::now();
    duration<double, std::milli> ms_double = t2 - t1;
    printf("%f seconds\n", ms_double.count() / 1e3);
    printf("Writing results: ");

    // Write Results
    {

        std::string outpath = sim.getResultPath();
        t1 = high_resolution_clock::now();
        writeResults(hostdx2, hostdx4, mdx2, mdx4, hostSimP, w_swc, iter, size, nrow, outpath);
        std::string allDataPath = outpath;
        if (SaveAll) {
            allDataPath.append("/allData.bin");
            FILE *outFile = fopen(allDataPath.c_str(), "wb");
            fwrite(hostAllData, SOD, iter * size * 3, outFile);
            fclose(outFile);
        }
        // write reflections and uref
        std::string reflectionsPath = outpath;
        reflectionsPath.append("/reflections.bin");
        FILE *outFile = fopen(reflectionsPath.c_str(), "wb");
        fwrite(hostReflections, SOD, iter * size * 3, outFile);
        fclose(outFile);
        std::string urefPath = outpath;
        urefPath.append("/uref.bin");
        outFile = fopen(urefPath.c_str(), "wb");
        fwrite(hosturef, SOD, iter * size * 3, outFile);
        fclose(outFile);

        // write sig0 and sigRe
        std::string sig0Path = outpath;
        sig0Path.append("/sig0.bin");
        outFile = fopen(sig0Path.c_str(), "wb");
        fwrite(hostSig0, SOD, timepoints, outFile);
        fclose(outFile);
        std::string sigRePath = outpath;
        sigRePath.append("/sigRe.bin");
        outFile = fopen(sigRePath.c_str(), "wb");
        fwrite(hostSigRe, SOD, Nbvec * iter, outFile);
        fclose(outFile);
    }

    t2 = high_resolution_clock::now();
    ms_double = t2 - t1;
    printf("%f seconds\n", ms_double.count() / 1e3);

    // Free Host Memory
    {

        free(hostdx2);
        free(hostdx4);
        free(hostSimP);
        free(hostD4Swc);
        free(hostNewIndex);
        free(mdx2);
        free(mdx4);
        free(hostAllData);

        // Reflection Variables
        free(hostReflections);
        free(hosturef);
        free(hostFlip);

        // Signal Variables
        free(hostT2);
        free(hostT);
        free(hostSigRe);
        free(hostSig0);
        free(hostbvec);
        free(hostbval);
        free(hostTD);
    }
    printf("Done!\n");
    return 0;
}
